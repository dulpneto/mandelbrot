#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/complex.h>
#include <sys/time.h>
#include <omp.h>
#include <hip/hip_runtime.h>

extern "C" {
  #include "write.h"
}

const int MAX_ITER = 100;

/*CPU*/
void runCPU(int c0_real, int c0_imag, int c1_real, int c1_imag, int width, int height, int threads, char *output);
void mandelbrotCPU(int index, int c0_real, int c0_imag, int c1_real, int c1_imag, int width, int height, int *buffer);

/*GPU*/
void runGPU(int c0_real, int c0_imag, int c1_real, int c1_imag, int width, int height, int threads, char *output);
__global__ void mandelbrotGPU(int *c0_real, int *c0_imag, int *c1_real, int *c1_imag, int *width, int *height, int *buffer);


int main(int argc, char *argv[]){
    //Input params
    if(argc < 10){
      printf("USAGE: mbrot <C0_REAL> <C0_IMAG> <C1_REAL> <C1_IMAG> <W> <H> <CPU/GPU> <THREADS> <OUTPUT>");
    }

    struct timeval start, end;

    int c0_real = atof(argv[1]);
    int c0_imag = atof(argv[2]);
    int c1_real = atof(argv[3]);
    int c1_imag = atof(argv[4]);
    int width = atof(argv[5]);
    int height = atof(argv[6]);
    char *execution = argv[7];
    int threads = atof(argv[8]);
    char *output = argv[9];

    gettimeofday(&start, NULL);

    if(strncmp(execution, "CPU", 3) == 0){
      runCPU(c0_real, c0_imag, c1_real, c1_imag, width, height, threads, output);
    } else {
      runGPU(c0_real, c0_imag, c1_real, c1_imag, width, height, threads, output);
    }

    gettimeofday(&end, NULL);
    double elapsed_time = (end.tv_sec - start.tv_sec) +
                              (end.tv_usec - start.tv_usec) / 1000000.0;
    printf("%.4fs\n", elapsed_time);

    return 0;
}

void runCPU(int c0_real, int c0_imag, int c1_real, int c1_imag, int width, int height, int threads, char *output){
  int *buffer = (int *) malloc(width * height * sizeof(int));

  omp_set_num_threads(threads);

  #pragma omp parallel for
  for (int i = 0; i < width * height; i++){
    mandelbrotCPU(i, c0_real, c0_imag, c1_real, c1_imag, width, height, buffer);
  }

  writeImage(output, width, height, MAX_ITER, buffer);

  free(buffer);
}

void mandelbrotCPU(int index, int c0_real, int c0_imag, int c1_real, int c1_imag, int width, int height, int *buffer){
    int i;
    int y = index / width;
    int x = index % width;
    float w = width;
    float h = height;

    thrust::complex<float> c = thrust::complex<float>((c0_real + (x / w) * (c1_real - c0_real)),
                                 (c0_imag + (y / h) * (c1_imag - c0_imag)));
    
    thrust::complex<float> z = thrust::complex<float>(0,0);

    for(i = 0; i < MAX_ITER; i++) {
        z = z*z + c;
        if(z.real() > 2 || z.imag() > 2) break;
    }
    
    buffer[y*width + x] = (i == MAX_ITER ? 0 : i);
}

void runGPU(int c0_real, int c0_imag, int c1_real, int c1_imag, int width, int height, int threads, char *output){
  int *buffer = (int *) malloc(width * height * sizeof(int));

  int *d_c0_real, *d_c0_imag, *d_c1_real, *d_c1_imag;
  int *d_width, *d_height;
  int *d_buffer;

  //cuda alloc
  hipMalloc(&d_c0_real, sizeof(int));
  hipMalloc(&d_c0_imag, sizeof(int));
  hipMalloc(&d_c1_real, sizeof(int));
  hipMalloc(&d_c1_imag, sizeof(int));
  hipMalloc(&d_width, sizeof(int));
  hipMalloc(&d_height, sizeof(int));
  hipMalloc((void **) &d_buffer, width * height * sizeof(int));
  //cuda alloc

  //memcpy
  hipMemcpy(d_c0_real, &c0_real, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_c0_imag, &c0_imag, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_c1_real, &c1_real, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_c1_imag, &c1_imag, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);
  //memcpy

  unsigned blocks_per_grid = ceil((width * height) / threads);
  mandelbrotGPU<<< blocks_per_grid , threads >>>(
    d_c0_real, d_c0_imag, d_c1_real, d_c1_imag, d_width, d_height, d_buffer);

  //sync
  hipDeviceSynchronize();
  hipMemcpy(buffer, d_buffer, width * height * sizeof(int), hipMemcpyDeviceToHost);

  //free cuda
  hipFree(d_c0_real);
  hipFree(d_c0_imag);
  hipFree(d_c1_real);
  hipFree(d_c1_imag);
  hipFree(d_width);
  hipFree(d_height);
  hipFree(d_buffer);
  //free cuda

  writeImage(output, width, height, MAX_ITER, buffer);

  free(buffer);
}

__global__ void mandelbrotGPU(int *c0_real, int *c0_imag, int *c1_real, int *c1_imag, int *width, int *height, int *buffer){
    

    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index > ((*width) * (*height)) - 1){
      return ;
    }
    int y = index / (*width);
    int x = index % (*width);
    int i;
    float w = *width;
    float h = *height;

    thrust::complex<float> c = thrust::complex<float>((*c0_real + (x / w) * (*c1_real - *c0_real)),
                                 (*c0_imag + (y / h) * (*c1_imag - *c0_imag)));
    
    thrust::complex<float> z = thrust::complex<float>(0,0);

    for(i = 0; i < MAX_ITER; i++) {
        z = z*z + c;
        if(z.real() > 2 || z.imag() > 2) break;
    }
    
    buffer[y * (*width) + x] = (i == MAX_ITER ? 0 : i);
}